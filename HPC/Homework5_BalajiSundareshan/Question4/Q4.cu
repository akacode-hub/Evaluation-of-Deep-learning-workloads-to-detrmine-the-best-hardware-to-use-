#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

double PI25DT = 3.141592653589793238462643;         /* 25-digit-PI*/

__global__ void fill_dart_count(int *in_dart_counts, int *num_darts_thread)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    long toss, num_toss_in;
    double x, y;
    
    hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

    for (toss = 0; toss < *num_darts_thread; toss++) {
		x = hiprand_uniform(&rng); // Random x position in [0,1]
		y = hiprand_uniform(&rng); // Random y position in [0,1]

        if(x*x + y*y <= 1.0){
            num_toss_in += 1;
        }
    }

    in_dart_counts[tid] = num_toss_in;

}

__global__ void count_darts(int *in_dart_counts, long *num_darts, int *size)
{
    int i;
    long sum = 0.0;
    for(i = 0; i<*size; i++){
        sum += in_dart_counts[i];
    }
    *num_darts = sum;    
}

int main(int argc, char *argv[])
{   
    srand(time(NULL));
    
    struct timespec start, end;

    assert(("./Q1 <number of darts>", argc == 2));
    long num_darts = atoi(argv[1]);
    int num_blocks = 1000;
    int num_threads_per_block = 256;
    assert(("num_darts should be divisible by 256*1000", num_darts % num_blocks * num_threads_per_block == 0));

    int num_dart_per_thread = num_darts / (num_blocks * num_threads_per_block);

    printf("Number of blocks: %d\n", num_blocks);
    printf("Number of threads per block: %d\n", num_threads_per_block);
    printf("Number of darts per thread: %d\n", num_dart_per_thread);
    printf("Total number of darts thrown: %d\n", num_darts);

    clock_gettime(CLOCK_MONOTONIC, &start);

    int *dart_counts_block;
    int *size_gpu, size;
    hipMalloc((void**)&dart_counts_block, num_blocks * num_threads_per_block * sizeof(int));
    hipMalloc((void**)&size_gpu, sizeof(size));

    long *num_darts_gpu, num_darts_in;
    hipMalloc((void**)&num_darts_gpu, sizeof(num_darts_in));

    int *num_dart_per_thread_gpu;
    hipMalloc((void**)&num_dart_per_thread_gpu, sizeof(num_dart_per_thread));

    fill_dart_count<<<num_blocks, num_threads_per_block>>>(dart_counts_block, num_dart_per_thread_gpu);

    count_darts<<<1, 1>>>(dart_counts_block, num_darts_gpu, size_gpu);

    hipMemcpy(&num_darts_in, num_darts_gpu, sizeof(num_darts_in), hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC, &end);
    double time_elapsed = (end.tv_sec - start.tv_sec);
    time_elapsed += (end.tv_nsec - start.tv_nsec) / 1000000000.0;

    long calculated_pi = (4*num_darts_in)/((double) num_darts);
    double error = fabs(calculated_pi - PI25DT);
    printf("Elapsed time = %f seconds \n", time_elapsed);
    printf("Calculated Pi is %.16f, Error is %.16f\n", calculated_pi, error);

    return 0;
}