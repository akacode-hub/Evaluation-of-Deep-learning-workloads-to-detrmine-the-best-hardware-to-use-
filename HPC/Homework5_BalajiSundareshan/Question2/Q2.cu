#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define MIN_NUM 1
#define MAX_NUM 100
const int N = 4;
const int num_threads_per_block = 2;
const int num_blocks = 2;
const int sblock_size = num_threads_per_block + 2;

__global__ void non_tile_compute(float b[][N][N], float a[][N][N])
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i > 0 && i < N-1 && j > 0 && j < N-1 && k > 0 &&  k < N-1)
    {
        a[i][j][k]=0.8*(b[i-1][j][k]+b[i+1][j][k]+b[i][j-1][k]
                        + b[i][j+1][k]+b[i][j][k-1]+b[i][j][k+1]);
    }

}

__global__ void tile_compute(float b[][N][N], float a[][N][N])
{   
    float __shared__ shared_b[sblock_size][sblock_size][sblock_size];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int offx, offy, offz;
    int offset = num_threads_per_block;

    for(offx=0; offx<sblock_size; offx+=2){
        for(offy=0; offy<sblock_size; offy+=2){
            for(offz=0; offz<sblock_size; offz+=2){
                shared_b[threadIdx.x + offx][threadIdx.y + offy][threadIdx.z + offz] = b[i][j][k];
                printf("i: %d, j: %d, k: %d, ox: %d, oy: %d, oz: %d\n", i, j, k, threadIdx.x + offx, threadIdx.y + offy, threadIdx.z + offz);
            }
        }
    }

    __syncthreads();

    if(blockIdx.x==0 && blockIdx.y==0 && blockIdx.z==0 && threadIdx.x==0 && threadIdx.y==0 && threadIdx.z==0){
    printf("shared: \n\n");
    int i_, j_, k_;
    for(i_=0;i_<sblock_size;i_++){
        for(j_=0;j_<sblock_size;j_++){
            for(k_=0;i_<sblock_size;k_++){
                printf("%f, ", shared_b[i_][j_][k_]);
                }
            }
        }
    }

    int tx = threadIdx.x + 1;
    int ty = threadIdx.y + 1;
    int tz = threadIdx.z + 1;

    if (i > 0 && i < N-1 && j > 0 && j < N-1 && k > 0 &&  k < N-1)
    {
        a[i][j][k] = 0.8*(shared_b[tx-1][ty][tz] + shared_b[tx+1][ty][tz] + shared_b[tx][ty-1][tz]
                        + shared_b[tx][ty+1][tz] + shared_b[tx][ty][tz-1] + shared_b[tx][ty][tz+1]);
    }

}

void gen_mat(float arr[][N][N]){

    int i, j, k;
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            for(k=0;k<N;k++){
                arr[i][j][k] = rand() % MAX_NUM + MIN_NUM;
                arr[i][j][k] = (float)rand()/(float)(RAND_MAX) * arr[i][j][k];
                //arr[i][j][k] = 1;
            }
        }
    }
}

void print_mat(float arr[][N][N]){

    int i, j, k;
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            for(k=0;k<N;k++){
                printf("%f, ", arr[i][j][k]);
            }
        }
    }
    printf("\n\n");
}

void compare_mat(float arr1[][N][N], float arr2[][N][N]){

    int i, j, k;
    
    for(k=0;k<N;k++){
        for(j=0;j<N;j++){
            for(i=0;i<N;i++){
                if (arr1[i][j][k] != arr2[i][j][k]){
                    printf("Test failed !!!\n");
                    return;
                }
            }
        }
    }
    printf("Test passed !!!\n");
}

int main(int argc, char *argv[])
{
    float * a_tile, * a_nontile, * b; 
    
    hipMallocManaged(&a_tile, N*N*N*sizeof(float));
    hipMallocManaged(&a_nontile, N*N*N*sizeof(float));
    hipMallocManaged(&b, N*N*N*sizeof(float));

    float b_vals[N][N][N];

    // generate data
    gen_mat(b_vals);

    // print data
    printf("Input matrix: \n");
    print_mat(b_vals);

    float (*a_vals_tile)[N][N] = reinterpret_cast<float (*)[N][N]>(a_tile);
    float (*a_vals_nontile)[N][N] = reinterpret_cast<float (*)[N][N]>(a_nontile);

    memcpy(b, &b_vals[0][0][0], sizeof(b_vals));

    dim3 threads_per_block(num_threads_per_block, num_threads_per_block, num_threads_per_block);
    dim3 blocks(num_blocks, num_blocks, num_blocks);

    non_tile_compute<<<blocks, threads_per_block>>>(reinterpret_cast<float (*)[N][N]>(b), a_vals_nontile);

    tile_compute<<<blocks, threads_per_block>>>(reinterpret_cast<float (*)[N][N]>(b), a_vals_tile);

    hipDeviceSynchronize();

    // print result
    compare_mat(a_vals_tile, a_vals_nontile);

    printf("Tiled: \n");
    print_mat(a_vals_tile);

    printf("Non Tiled: \n");
    print_mat(a_vals_nontile);

    return 0;
}




// __syncthreads();