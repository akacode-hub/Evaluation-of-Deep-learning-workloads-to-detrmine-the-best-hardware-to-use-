#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define MIN_NUM 1
#define MAX_NUM 1000000

const int num_classes = 2;
const int num_blocks = 1;
const int num_threads_per_block = 1;
int tot_threads = num_blocks * num_threads_per_block;

__device__ int find_bin(int val, float * min_range_cls, float * max_range_cls){

    int i;
    
    for(i = 0; i < num_classes; i++) 
    {
        if(val>=min_range_cls[i] && val<max_range_cls[i])
            return  i;
    }

    return -1;
}

__global__ void hist_binning(int * data, int * hist_bin, int * cls_el, float * min_range_cls, float * max_range_cls, int * min_tidxs, int * max_tidxs)
{   
    int __shared__ hist_per_block[num_classes];

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    
    int min_id, max_id;
    int i, cls_id;

    for (i = 0; i < num_classes; i += 1) 
        hist_per_block[i] = 0;

    __syncthreads();

    min_id = min_tidxs[tid];
    max_id = max_tidxs[tid];
    // printf("tid: %d, min_id: %d, max_id: %d\n", tid, min_id, max_id);

    for(i=min_id;i<max_id;i++){
        cls_id = find_bin(data[i], min_range_cls, max_range_cls);    
        hist_per_block[cls_id] += 1;
        cls_el[cls_id] = data[i];
        //printf("1 i: %d, val: %d, cls_id: %d\n", i, data[i], cls_id);
    }

    __syncthreads();
    
    for (i = 0; i < num_classes; i += 1) 
    {   
        hist_bin[i] += hist_per_block[i];
        printf("bid: %d, bblk: %d, tid: %d, hist %d: %d\n", blockIdx.x, blockDim.x, threadIdx.x, i, hist_bin[i]);
    }
}

void dist_data_tids(int * min_tidxs, int * max_tidxs, int data_len)
{

    int i;
    float interval = (float) data_len / (float) tot_threads;
    interval = floor(interval);

    for(i=0; i<tot_threads; i++){

        min_tidxs[i] = i*interval;
        max_tidxs[i] = (i+1)*interval;

        if(i == tot_threads-1){
            max_tidxs[i] = data_len;
        }

    }
}

void gen_data(int * data, int data_len)
{

    int i;
    for(i=0; i<data_len; i++){
        data[i] = rand() % MAX_NUM + MIN_NUM;
    }

}

void set_classes(float * min_range_cls, float * max_range_cls, int num_classes){
    
    int i;
    
    float range = MAX_NUM - MIN_NUM;
    float interval = (float) range / (float) num_classes;
    interval = ceil(interval);
    
    for(i=0; i<num_classes; i++){

        min_range_cls[i] = i*interval + MIN_NUM;

        if(i == num_classes-1){
            max_range_cls[i] = MAX_NUM + 1;
        }else{
            max_range_cls[i] = (i+1)*interval + MIN_NUM;
        }
    }
}

void print_data(int * data, float * min_range_cls, float * max_range_cls, int * min_tids, int * max_tids, int data_len, int num_classes){

    int i;

    // printf("Input data: \n");
    // for(i=0; i<data_len; i++){
    //     printf("%d ",data[i]);
    // }
    // printf("\n");

    printf("Class range: \n");
    for(i=0; i<num_classes; i++){
        printf("Class %d Min: %f, Max: %f\n", i, min_range_cls[i], max_range_cls[i]);
    }

    printf("Data ID range: \n");
    for(i=0; i<tot_threads; i++){
        printf("data %d Min: %d, Max: %d\n", i, min_tids[i], max_tids[i]);
    }

}

void print_results(int * hist_data, int * cls_el){

    int i;
    int sum = 0;
    for(i=0;i<num_classes;i++){
        sum += hist_data[i];
        printf("number of samples in cls %d: %d\n", i, hist_data[i]);
    }
    printf("histogram sum: %d\n", sum);

    for(i=0;i<num_classes;i++){
        printf("One element from cls %d: %d\n", i, cls_el[i]);
    }

}

int main(int argc, char *argv[])
{   
    srand(time(NULL));
    struct timespec start, end;
    int data_len;
    int * data, * data_gpu;
    int * data_cls_map, * data_cls_map_gpu;

    int * min_tidxs, * max_tidxs;
    float * min_range_cls, * max_range_cls;
    int * min_tidxs_gpu, * max_tidxs_gpu;
    float * min_range_cls_gpu, * max_range_cls_gpu;

    int * hist_bin, * hist_bin_gpu;
    int * cls_el, * cls_el_gpu;

    assert(("./Q1 <number of values>", argc == 2));

    data_len = atoi(argv[1]);

    min_range_cls = (float *)calloc(num_classes, sizeof(float));
    max_range_cls = (float *)calloc(num_classes, sizeof(float));
    hipMalloc((void **) &min_range_cls_gpu, sizeof(float)*num_classes);
    hipMalloc((void **) &max_range_cls_gpu, sizeof(float)*num_classes);

    min_tidxs = (int *)calloc(tot_threads, sizeof(int));
    max_tidxs = (int *)calloc(tot_threads, sizeof(int));
    hipMalloc((void **) &min_tidxs_gpu, sizeof(int)*tot_threads);
    hipMalloc((void **) &max_tidxs_gpu, sizeof(int)*tot_threads);

    hist_bin = (int *)calloc(num_classes, sizeof(int));
    hipMalloc((void **) &hist_bin_gpu, sizeof(int)*num_classes);
    hipMemset(hist_bin_gpu, 0, sizeof(int)*num_classes);

    cls_el = (int *)calloc(num_classes, sizeof(int));
    hipMalloc((void **) &cls_el_gpu, sizeof(int)*num_classes);

    data = (int *)calloc(data_len, sizeof(int));
    gen_data(data, data_len);
    set_classes(min_range_cls, max_range_cls, num_classes);
    dist_data_tids(min_tidxs, max_tidxs, data_len);

    print_data(data, min_range_cls, max_range_cls, min_tidxs, max_tidxs, data_len, num_classes);

    hipMalloc((void **) &data_gpu, sizeof(int)*data_len);
    
    hipMemcpy(data_gpu, data, sizeof(int)*data_len, hipMemcpyHostToDevice);   

    hipMemcpy(min_range_cls_gpu, min_range_cls, sizeof(int)*num_classes, hipMemcpyHostToDevice);
    hipMemcpy(max_range_cls_gpu, max_range_cls, sizeof(int)*num_classes, hipMemcpyHostToDevice);

    hipMemcpy(min_tidxs_gpu, min_tidxs, sizeof(int)*tot_threads, hipMemcpyHostToDevice);
    hipMemcpy(max_tidxs_gpu, max_tidxs, sizeof(int)*tot_threads, hipMemcpyHostToDevice);

    hist_binning<<<num_blocks, num_threads_per_block>>>(data_gpu, hist_bin_gpu, cls_el_gpu, min_range_cls_gpu, max_range_cls_gpu, min_tidxs_gpu, max_tidxs_gpu);

    hipMemcpy(hist_bin, hist_bin_gpu, sizeof(int)*num_classes, hipMemcpyDeviceToHost);
    hipMemcpy(cls_el, cls_el_gpu, sizeof(int)*num_classes, hipMemcpyDeviceToHost);

    print_results(hist_bin, cls_el);

    hipFree(data_gpu); 
    hipFree(min_range_cls_gpu);
    hipFree(max_range_cls_gpu);
    hipFree(min_tidxs_gpu);
    hipFree(max_tidxs_gpu);
    hipFree(hist_bin_gpu);
    hipFree(cls_el_gpu);
}
